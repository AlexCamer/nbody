#include "hip/hip_runtime.h"
#include "constants.cuh"
#include "physics.cuh"

namespace nbody
{
	void
	updateAcc(const float4* posArray_host, float4* accArray_host, const float* massArray_host,
		size_t size, unsigned int bodyIdx)
	{
		// retrieve data for myBody (body assigned to block)
		float4 myBodyPos = posArray_host[bodyIdx];
		float4 myBodyAcc = { 0.0f, 0.0f, 0.0f };

		// calculate acceleration of myBody due to all other bodies
		for (unsigned int i = 0; i < size; i++) {
			float4 otherBodyPos = posArray_host[i];
			float otherBodyMass = massArray_host[i];

			// calculate distance vector between myBody and otherBody
			float4 dist;
			dist.x = otherBodyPos.x - myBodyPos.x;
			dist.y = otherBodyPos.y - myBodyPos.y;
			dist.z = otherBodyPos.z - myBodyPos.z;

			// calculate ratio between acceleration and distance
			float temp = dist.x * dist.x + dist.y * dist.y + dist.z * dist.z + NAN_GUARD;
			temp = temp * temp * temp;
			float ratio = otherBodyMass / sqrtf(temp);

			// update myBody acceleration
			myBodyAcc.x += dist.x * ratio;
			myBodyAcc.y += dist.y * ratio;
			myBodyAcc.z += dist.z * ratio;
		}

		// store myBody acceleration
		accArray_host[bodyIdx] = myBodyAcc;
	}

	void
	updatePosAndVel(float4* posArray_host, float4* velArray_host, const float4* accArray_host,
		float dt, unsigned int bodyIdx)
	{
		// retrieve data for myBody (body assigned to block)
		float4 myBodyPos = posArray_host[bodyIdx];
		float4 myBodyVel = velArray_host[bodyIdx];
		float4 myBodyAcc = accArray_host[bodyIdx];

		// update myBody position
		float dtHalfSqr = dt * dt / 2;
		myBodyPos.x += myBodyVel.x * dt + myBodyAcc.x * dtHalfSqr;
		myBodyPos.y += myBodyVel.y * dt + myBodyAcc.y * dtHalfSqr;
		myBodyPos.z += myBodyVel.z * dt + myBodyAcc.z * dtHalfSqr;

		// update myBody velocity
		myBodyVel.x += myBodyAcc.x * dt;
		myBodyVel.y += myBodyAcc.y * dt;
		myBodyVel.z += myBodyAcc.z * dt;

		// store myBody position and velocity
		posArray_host[bodyIdx] = myBodyPos;
		velArray_host[bodyIdx] = myBodyVel;
	}
}
