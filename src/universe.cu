#include "hip/hip_runtime.h"
#include <stdio.h>
#include "constants.cuh"
#include "physics.cuh"
#include "universe.cuh"

__host__ struct universe *
universe_create(const float3 *pos_host, const float3 *vel_host, const float *mass_host, unsigned int n)
{
	hipError_t cuda_status;

	// allocate memory (on host) for universe struct
	struct universe *univ;
	cuda_status = hipHostMalloc((void **)&univ, sizeof(struct universe));
	if (univ == NULL) {
		fprintf(stderr, "Failed to allocate memory (on host) for universe struct.");
		return NULL;
	}

	univ->pos_dev = NULL;
	univ->vel_dev = NULL;
	univ->acc_dev = NULL;
	univ->mass_dev = NULL;
	univ->n = n;

	// allocate memory on device for position vector
	cuda_status = hipMalloc((void **)&univ->pos_dev, n * sizeof(float3));
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory on device for position vector.");
		goto error;
	}

	// allocate memory on device for velocity vector
	cuda_status = hipMalloc((void **)&univ->vel_dev, n * sizeof(float3));
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory on device for velocity vector.");
		goto error;
	}

	// allocate memory on device for acceleration vector
	cuda_status = hipMalloc((void **)&univ->acc_dev, n * sizeof(float3));
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory on device for acceleration vector.");
		goto error;
	}

	// allocate memory on device for mass vector
	cuda_status = hipMalloc((void **)&univ->mass_dev, n * sizeof(float));
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to allocate memory on device for mass vector.");
		goto error;
	}

	// copy position vector from host to device
	cuda_status = hipMemcpy(univ->pos_dev, pos_host, n * sizeof(float3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to copy position vector from host to device.");
		goto error;
	}

	// copy velocity vector from host to device
	cuda_status = hipMemcpy(univ->vel_dev, vel_host, n * sizeof(float3), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to copy velocity vector from host to device.");
		goto error;
	}

	// copy mass vector from host to device
	cuda_status = hipMemcpy(univ->mass_dev, mass_host, n * sizeof(float), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to copy mass vector from host to device.");
		goto error;
	}

	return univ;

error:
	// destroy universe struct and vectors
	universe_destroy(univ);
	return NULL;
}

__host__ void
universe_destroy(struct universe *univ)
{
	hipFree(univ->pos_dev);
	hipFree(univ->vel_dev);
	hipFree(univ->acc_dev);
	hipFree(univ->mass_dev);
	hipHostFree(univ);
}

__host__ int
universe_step(struct universe *univ)
{
	hipError_t cuda_status;

	// update acceleration vector based on position and mass vectors
	update_acc<<<univ->n, 1>>>(univ->pos_dev, univ->acc_dev, univ->mass_dev, univ->n);
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to update acceleration vector.");
		return 1;
	}

	// wait for device to finish computation
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to synchronize host and device.");
		return 1;
	}

	// update position and velocity vectors based on the new acceleration vector
	update_pos_and_vel<<<univ->n, 1>>>(univ->pos_dev, univ->vel_dev, univ->acc_dev);
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to update position and velocity vectors.");
		return 1;
	}

	// wait for device to finish computation
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to synchronize host and device.");
		return 1;
	}

	return 0;
}

__host__ int
universe_state(const struct universe *univ, float3 *pos_host)
{
	// copy position vector from host to device
	hipError_t cuda_status = hipMemcpy(pos_host, univ->pos_dev, univ->n * sizeof(float3), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Failed to copy position vector from device to host.");
		return 1;
	}
	return 0;
}
